#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C++"
{
#include "glife.h"
}
#include <hip/hip_runtime.h>

// HINT: YOU CAN USE THIS METHOD FOR ERROR CHECKING
// Print error message on CUDA API or kernel launch
#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
        }                                                      \
    } while (0);

__device__ const int liveTable[2][10] = {{0, 0, 0, 1, 0, 0, 0, 0, 0, 0}, {0, 0, 0, 1, 1, 0, 0, 0, 0, 0}};
// TODO: YOU MAY NEED TO USE IT OR CREATE MORE
__device__ int getNeighbors(int *grid, int tot_rows, int tot_cols,
                            int rows, int cols)
{
    int numOfNeighbors = 0;
    int row = rows - 1 < 0 ? 0 : rows - 1;
    int rows_max = rows + 1 == tot_rows ? rows : rows + 1;
    int cols_max = cols + 1 == tot_cols ? cols : cols + 1;

    for (; row <= rows_max; row++)
    {
        int col = cols - 1 < 0 ? 0 : cols - 1;
        for (; col <= cols_max; col++)
            numOfNeighbors += grid[row * tot_cols + col];
    }
    return numOfNeighbors;
}

// TODO: YOU NEED TO IMPLEMENT KERNEL TO RUN ON GPU DEVICE
__global__ void kernel(int *grid, int *temp, int rows, int cols)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int num;
    if (index < rows * cols)
    {
        num = getNeighbors(grid, rows, cols, index / cols, index % cols);
        //temp[index] = liveTable[grid[index]][num];
        
        if (num == 3 || (grid[index] && num == 2))
            temp[index] = 1;
        else
            temp[index] = 0;
    }
}
__global__ void swap(int *grid, int *temp, int rows, int cols)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < rows * cols)
    {
        grid[index] = temp[index];
        temp[index] = 0;
    }
}

// TODO: YOU NEED TO IMPLEMENT TO PRINT THE INDEX RESULTS
void cuda_dump()
{
    printf("===============================\n");

    printf("===============================\n");
}

// TODO: YOU NEED TO IMPLEMENT TO PRINT THE INDEX RESULTS
void cuda_dump_index()
{
    printf(":: Dump Row Column indices\n");
}

// TODO: YOU NEED TO IMPLEMENT ON CUDA VERSION
uint64_t runCUDA(int rows, int cols, int gen,
                 GameOfLifeGrid *g_GameOfLifeGrid, int display)
{
    hipSetDevice(0); // DO NOT CHANGE THIS LINE

    uint64_t difft;

    // Start timer for CUDA kernel execution
    difft = dtime_usec(0);
    // ---------- TODO: CALL CUDA API HERE ----------


    int size = sizeof(int) * (g_GameOfLifeGrid->getRows() * g_GameOfLifeGrid->getCols());
    int *d_Grid = NULL;
    hipMalloc((void **)&d_Grid, size);
    int *d_Temp = NULL;
    hipMalloc((void **)&d_Temp, size);

    hipMemcpy(d_Grid, *(g_GameOfLifeGrid->getGrid()), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    while (gen--)
    {
        kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Grid, d_Temp, rows, cols);
        swap<<<blocksPerGrid, threadsPerBlock>>>(d_Grid, d_Temp, rows, cols);
    }


    
    hipMemcpy(*(g_GameOfLifeGrid->getGrid()), d_Grid, size, hipMemcpyDeviceToHost);

    hipFree(d_Grid);
    hipFree(d_Temp);
    // Finish timer for CUDA kernel execution
    difft = dtime_usec(difft);

    // Print the results
    if (display)
    {
        g_GameOfLifeGrid->dump();
        g_GameOfLifeGrid->dumpIndex();
    }
    return difft;
}
#endif
